#include<iostream>
#include<hip/hip_runtime.h>
#include<math.h>
#include <time.h>


/* 1-  nvcc  acopladas_B3-2.cu -o acopladas_B3-2
  
   2-./acopladas_B3-2

We are using Dormand-Prince Method based on  http://depa.fquim.unam.mx/amyd/archivero/DormandPrince_19856.pdf   
*/


using namespace std;

__global__  void suma(int *a,int *b,int *c)
{
  int id=threadIdx.x;
  c[id]=a[id]+b[id];
};



__global__  void  Resuelve(double *dev_w, double *dev_Ppo, double *dev_PsL_Ppo,double *dev_K,double *dev_Delta,double *dev_PpF,double *dev_PsF)
{
 int id=threadIdx.x;
 double x0[2];double y1[3],y2[3];
 double eps=0.000001;
 double t0=0;
    double h=0.01;
    double hmin=0.00001;
    double hmax=0.1;
    
    double gB=4e-14;
    double Aeff=85e-18;
    double a=0.2/4.343;
    double b=gB/(1.2*Aeff);
    double K=0.53;
    
    double PsL_Ppo=0.8;
    
    int c=0;
    
       
  while (PsL_Ppo>0.0000011)
 {
        dev_K[id]=dev_K[id]-dev_Delta[id];       
       // K=K-0.000001;
       K=dev_K[id];
       x0[0]=dev_Ppo[id];                   //x0[0]=0.0005;
       x0[1]=K*x0[0];                     //x0[1]=dev_K[id]*dev_Ppo[id];        
       
     
     
     y1[0]=x0[0]; y1[1]=x0[0];
     y2[0]=x0[1];  y2[1]=x0[1];
     c=c+1;
     t0=0;
     h=0.01;
     
     while(t0<=25)
    {
        ///////////////
        
        
         double k11=-(a)*y1[0]-(b)*y1[0]*y2[0];
         double k12=(a)*y2[0]-(b)*y1[0]*y2[0];
         k11=k11*h;
         k12=k12*h;
         
         
         double ty1_k11=y1[0]+(k11)*(1.0/5);
         double ty2_k12=y2[0]+(k12)*(1.0/5);
         
         double k21=-(a)*ty1_k11-(b)*ty1_k11*ty2_k12;
         double k22=(a)*ty2_k12-(b)*ty1_k11*ty2_k12;
         
         k21=k21*h;
         k22=k22*h;
         
         
         double ty1_k21=y1[0]+(3.0/40)*(k11)+(9.0/40)*(k21);
         double ty2_k22=y2[0]+(3.0/40)*(k12)+(9.0/40)*(k22);
         
         double k31=-(a)*ty1_k21-(b)*ty1_k21*ty2_k22;
         double k32=(a)*ty2_k22-(b)*ty1_k21*ty2_k22;
         
         k31=k31*h;
         k32=k32*h;
         
         
         double ty1_k31=y1[0]+(44.0/45)*k11-(56.0/15)*k21+(32.0/9)*(k31);
         double ty2_k32=y2[0]+(44.0/45)*k12-(56.0/15)*k22+(32.0/9)*(k32);
         
         
         double k41=-(a)*ty1_k31-(b)*ty1_k31*ty2_k32;
         double k42=(a)*ty2_k32-(b)*ty1_k31*ty2_k32;
         
         
         k41=k41*h;
         k42=k42*h;
         
         
         double ty1_k41=y1[0]+(19372.0/6561)*k11-(25360.0/2187)*k21+(64448.0/6561)*(k31)-(212.0/729)*(k41);
         double ty2_k42=y2[0]+(19372.0/6561)*k12-(25360.0/2187)*k22+(64448.0/6561)*(k32)-(212.0/729)*(k42);
         
         double k51=-(a)*ty1_k41-(b)*ty1_k41*ty2_k42;
         double k52=(a)*ty2_k42-(b)*ty1_k41*ty2_k42;
         
         k51=k51*h;
         k52=k52*h;
         
         double ty1_k51=y1[0]+(9017.0/3168)*k11-(355.0/33)*k21-(46732.0/5247)*(k31)+(49.0/176)*(k41)-(5103.0/18656)*(k51);
         double ty2_k52=y2[0]+(9017.0/3168)*k12-(355.0/33)*k22-(46732.0/5247)*(k32)+(49.0/176)*(k42)-(5103.0/18656)*(k52);
         
         double k61=-(a)*ty1_k51-(b)*ty1_k51*ty2_k52;
         double k62=(a)*ty2_k52-(b)*ty1_k51*ty2_k52;
         
         k61=k61*h;
         k62=k62*h;
         
         
         double ty1_k61=y1[0]+(35.0/384)*k11+(500.0/1113)*(k31)+(125.0/192)*(k41)-(2187.0/6784)*(k51)+(11.0/84)*(k61);
         double ty2_k62=y2[0]+(35.0/384)*k12+(500.0/1113)*(k32)+(125.0/192)*(k42)-(2187.0/6784)*(k52)+(11.0/84)*(k62);
         
         double k71=-(a)*ty1_k61-(b)*ty1_k61*ty2_k62;
         double k72=(a)*ty2_k62-(b)*ty1_k61*ty2_k62;
         
         k71=k71*h;
         k72=k72*h;
         
         double tmpy1=y1[0]+((35.0/384)*k11+(500.0/1113)*k31+(125.0/192)*k41-(2187.0/6784)*k51+(11.0/84)*k61);
         double tmpy2=y2[0]+((35.0/384)*k12+(500.0/1113)*k32+(125.0/192)*k42-(2187.0/6784)*k52+(11.0/84)*k62);
         
         // double tmpz1=y1[0]+(5179.0/57600)*k11+(7571.0/16695)*k31+(393.0/640)*k41-(92097.0/339200)*k51+(187.0/2100)*k61+(1.0/40)*k71;
         double tmpz2=y2[0]+(5179.0/57600)*k12+(7571.0/16695)*k32+(393.0/640)*k42-(92097.0/339200)*k52+(187.0/2100)*k62+(1.0/40)*k72;
         
         double err=abs(tmpy2-tmpz2);
         
         double s=pow((eps*h)/(2.0*err),1.0/5);
         
         
         double h1=s*h;
         
         if (h1<hmin)
         h1=hmin;
         else if(h1>hmax) h1=hmax;
         
         
         t0=t0+h;
         y1[0]=tmpy1;
         y2[0]=tmpy2;
         h=h1;
        
        ///////////////
        
        
      
    }  //fin del for
    
     double PsL=y2[0];
      //dev_PsL_Ppo[id]=double(PsL)/dev_Ppo[id]; 
       PsL_Ppo=double(PsL)/dev_Ppo[id];
       
} //fin del while tolerancia
 
 
 dev_PpF[id]=y1[0]; 
 dev_PsF[id]=y2[1];
  
 //dev_PpF[id]=10*log10(y1[0]/1e-3);
 //dev_PsF[id]=10*log10(y2[1]/1e-3)  ; 
   
   
 //dev_PpF[id]=8; 
 //dev_PsF[id]=9;
   
 
};



int main()
{
  double DELTA;
  double *host_w,*host_Ppo,*host_PpF,*host_PsF,*host_PsL_Ppo,*host_K,*host_Delta;
  double *dev_w,*dev_Ppo,*dev_PpF,*dev_PsF,*dev_PsL_Ppo,*dev_K,*dev_Delta;
  
  
  DELTA=0.000001;
  
  double incrementow=12.5;
  double iniciow=0.0;
  double finalw=25.0;
  int Nw=ceil((finalw-iniciow)/incrementow); 
  
  
  host_w=new double[Nw];
  Nw=Nw+1;
  
   for (int i=0;i<Nw;i++)
   {
      
      host_w[i]=iniciow;
      iniciow=iniciow+incrementow;
  };
  
  
  
  
  double incrementoPpo=0.0001;
  double inicioPpo=0.0005;
  double finalPpo=0.0045;
  int N=ceil((finalPpo-inicioPpo)/incrementoPpo); 
  N=N+1;
  host_Ppo=new double[N];  
  
  host_PsL_Ppo=new double[N];  
  host_K= new double[N];
  host_Delta=new double[N];
  host_PpF=new double[N];
  host_PsF=new double[N];
  
  
  
  for (int i=0;i<N;i++)
   {
      
      host_PpF[i]=0.0;
      host_PsF[i]=0.0;
      host_PsL_Ppo[i]=0.8;
      host_K[i]=0.53;
      host_Delta[i]=DELTA;
      host_Ppo[i]=inicioPpo;
      inicioPpo=inicioPpo+incrementoPpo;
  };
  
  
  int memw=sizeof(double)*Nw;
  int mem=sizeof(double)*N;
  
  hipMalloc((void **)&dev_Ppo,mem);
  hipMalloc((void **)&dev_w,memw);
  hipMalloc((void **)&dev_PsL_Ppo,mem);
  hipMalloc((void **)&dev_K,mem);
  hipMalloc((void **)&dev_Delta,mem);  
  hipMalloc((void **)&dev_PpF,mem); 
  hipMalloc((void **)&dev_PsF,mem);
  
  
  hipMemcpy(dev_w,host_w,memw,hipMemcpyHostToDevice);
  hipMemcpy(dev_Ppo,host_Ppo,mem,hipMemcpyHostToDevice); 
  hipMemcpy(dev_PsL_Ppo,host_PsL_Ppo,mem,hipMemcpyHostToDevice);
  hipMemcpy(dev_K,host_K,mem,hipMemcpyHostToDevice);
  hipMemcpy(dev_Delta,host_Delta,mem,hipMemcpyHostToDevice);
  hipMemcpy(dev_PpF,host_PpF,mem,hipMemcpyHostToDevice); 
  hipMemcpy(dev_PsF,host_PsF,mem,hipMemcpyHostToDevice); 
  
 
  hipEvent_t start,stop;
   float time;
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);
  
  
  Resuelve<<<1,N>>>(dev_w,dev_Ppo,dev_PsL_Ppo,dev_K,dev_Delta,dev_PpF,dev_PsF);
  
  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&time,start,stop);
  cout<<"tiempo : "<<time/1000<<endl;
  printf("\nelapsedTime in ms= %f:\n",time);
  
  hipMemcpy(host_K,dev_K,mem,hipMemcpyDeviceToHost);
  hipMemcpy(host_PsL_Ppo,dev_PsL_Ppo,mem,hipMemcpyDeviceToHost);
  hipMemcpy(host_PpF,dev_PpF,mem,hipMemcpyDeviceToHost);
  hipMemcpy(host_PsF,dev_PsF,mem,hipMemcpyDeviceToHost);
  

  cout<<endl;
  cout<<N;
  cout<<endl;
  
  cout<<"PpF"<<endl;
  for(int i=0;i<N;i++)
  {
     cout<<" "<<host_PpF[i]<<" ";
  };
  
  cout<<endl;
  cout<<N;
  cout<<endl;
    cout<<"PsF"<<endl;
  
  for(int i=0;i<N;i++)
  {
     cout<<" "<<host_PsF[i]<<" ";
  };
  
  cout<<endl;
  
  hipFree(dev_Ppo);
  hipFree(dev_w);
  hipFree(dev_K);
  hipFree(dev_PsL_Ppo);
  hipFree(dev_Delta);
  hipFree(dev_PpF);
  hipFree(dev_PsF);
  
  
  
 
 
  delete [] host_Ppo;
  delete [] host_w; 
  delete [] host_K; 
  delete [] host_PsL_Ppo; 
  delete [] host_Delta;
  delete [] host_PpF;
  delete [] host_PsF;
  
  
};

